#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include ""
//#include <iostream>
// 
//using namespace std;
// 
//int main()
//{
//	hipDeviceProp_t deviceProp;
//	int deviceCount;
//	hipError_t hipError_t;
//	hipError_t = hipGetDeviceCount(&deviceCount);
//	for (int i = 0; i < deviceCount; i++)
//	{
//		hipError_t = hipGetDeviceProperties(&deviceProp, i);
// 
//		cout << "�豸 " << i + 1 << " ����Ҫ���ԣ� " << endl;
//		cout << "�豸�Կ��ͺţ� " << deviceProp.name << endl;
//		cout << "�豸ȫ���ڴ���������MBΪ��λ���� " << deviceProp.totalGlobalMem / 1024 / 1024 << endl;
//		cout << "�豸��һ���߳̿飨Block���п��õ�������ڴ棨��KBΪ��λ���� " << deviceProp.sharedMemPerBlock / 1024 << endl;
//		cout << "�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� " << deviceProp.regsPerBlock << endl;
//		cout << "�豸��һ���߳̿飨Block���ɰ���������߳������� " << deviceProp.maxThreadsPerBlock << endl;
//		cout << "�豸�ļ��㹦�ܼ���Compute Capability���İ汾�ţ� " << deviceProp.major << "." << deviceProp.minor << endl;
//		cout << "�豸�϶ദ������������ " << deviceProp.multiProcessorCount << endl;
//	}
//	getchar();
//	return 0;
//}
/**
 *   addition: C = A + B.
 *
 * This samp le is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */
#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define NX 512
#define NY 512
#define NZ 512

 
using namespace std;
 
void main()
{
	int n = NX;  // 100
	float2 *data_h = new float2[n]; // ����������� ע����float2����
	for (int i=0; i<n; ++i)
	{
		data_h[i].x = 1;
		data_h[i].y = i;
	}
	float2 *data_d;
	hipMalloc((void**)&data_d, n * sizeof(float2));  //�����Դ�ռ�
	hipMemcpy(data_d, data_h, n * sizeof(float2), hipMemcpyHostToDevice);  //�����ݿ������豸��
 
	static StopWatchInterface *timer;      // ���ڲ���ʱ��ĺ���
	sdkCreateTimer(&timer);   // ��ʼ��ʱ��
 
	sdkStartTimer(&timer);  // ��ʼ��ʱ
 
	// ����CUFFT���
	hipfftHandle plan1;
	hipfftPlan1d(&plan1, n, HIPFFT_C2C, 1);
 
	sdkStopTimer(&timer);  // ��ʱ����
	cout<<"Create plan1: "<<sdkGetTimerValue(&timer)<<"ms"<<endl;
 
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
 
	hipfftExecC2C(plan1, data_d, data_d, HIPFFT_FORWARD);  // ���ٸ���Ҷ���仯
	hipfftDestroy(plan1);
 
	sdkStopTimer(&timer);
	cout<<"Execute plan1: "<<sdkGetTimerValue(&timer)<<"ms"<<endl;
 
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
 
	hipfftHandle plan2;
	hipfftPlan1d(&plan2, n, HIPFFT_C2C, 1);
 
	sdkStopTimer(&timer);
	cout<<"Create plan2: "<<sdkGetTimerValue(&timer)<<"ms"<<endl;
 
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
 
	hipfftExecC2C(plan2, data_d, data_d, HIPFFT_BACKWARD);  // ���ٸ���Ҷ��仯
	hipfftDestroy(plan2);
 
	sdkStopTimer(&timer);
	cout<<"Execute plan2: "<<sdkGetTimerValue(&timer)<<"ms"<<endl;

	hipFree(data_d);
	delete data_h;
	

	//---------------------------- ��ά����Ҷ�任

    n = NX*NY;
	float2 *idata_h = new float2[n]; // ����������� ע����float2����
	for (int i=0; i<n; ++i)
	{
		idata_h[i].x = i;
		idata_h[i].y = i;
	}
	float2 *idata_d,*odata_d;
	hipMalloc((void**)&idata_d, n * sizeof(float2));  //�����Դ�ռ�
	hipMalloc((void**)&odata_d, n * sizeof(float2)); 
	hipMemcpy(idata_d, idata_h, n * sizeof(float2), hipMemcpyHostToDevice);  //�����ݿ������豸��
 
	sdkResetTimer(&timer);   // ��ʼ��ʱ��
	sdkStartTimer(&timer);  // ��ʼ��ʱ

	// ����CUFFT���
	hipfftHandle plan3;
	hipfftPlan2d(&plan3, NX, NY, HIPFFT_C2C);

	// ִ��CUFFT
	hipfftExecC2C(plan3, idata_d, odata_d, HIPFFT_FORWARD);  // ���ٸ���Ҷ��仯
		
	sdkStopTimer(&timer);
	cout<<"Execute plan3(2D_FFT): "<<sdkGetTimerValue(&timer)<<"ms"<<endl;

	hipfftDestroy(plan3);
	hipFree(idata_d);
	hipFree(odata_d);
	delete idata_h;


	//--------------------��άFFT
	n = NX*NY*NZ;
	float2 *idata_h3 = new float2[n]; // ����������� ע����float2����
	for (int i=0; i<n; ++i)
	{
		idata_h3[i].x = 1;
		idata_h3[i].y = 1;
	}
	float2 *idata_d3,*odata_d3;
	hipMalloc((void**)&idata_d3, n * sizeof(float2));  //�����Դ�ռ�
	hipMalloc((void**)&odata_d3, n * sizeof(float2)); 
	hipMemcpy(idata_d3,idata_h3, n * sizeof(float2), hipMemcpyHostToDevice);  //�����ݿ������豸��
 
	sdkResetTimer(&timer);   // ��ʼ��ʱ��
	sdkStartTimer(&timer);  // ��ʼ��ʱ

	// ����CUFFT���
	hipfftHandle plan4;
	hipfftPlan3d(&plan4, NX, NY, NZ, HIPFFT_C2C);

	// ִ��CUFFT
	hipfftExecC2C(plan4, idata_d3, odata_d3, HIPFFT_FORWARD);  // ���ٸ���Ҷ���仯  512*512*512 ��ʱ2.17ms
		
	sdkStopTimer(&timer);
	cout<<"Execute plan4(3D_FFT): "<<sdkGetTimerValue(&timer)<<"ms"<<endl;

	hipMemcpy(idata_h3, odata_d3,  n * sizeof(float2),hipMemcpyDeviceToHost);


	hipfftDestroy(plan4);
	hipFree(idata_d3);
	hipFree(odata_d3);
	delete idata_h3;






	system("pause");
 
}
